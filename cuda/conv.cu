#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <sys/mman.h>

#define blockx 16
#define blocky 16

#include "conv.h"
#include "tictoc.h"
#include "iaca.h"


#define get(X,Y) inData[(Y*width)+X]
#define set(X,Y,V) outData[(Y*width)+X] = V

__global__ void kernel_conv3(float * inData, float *outData, unsigned long width, unsigned long height, const float * filter) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;

    float acc1, acc2, acc3;
    acc1 = 0;

    if (idx < width - 1 && idy < height - 1 && idx > 0 && idy > 0) {
        //Valid target
        acc1 = get(idx - 1, idy - 1) * filter[0];
        acc2 = get(idx, idy - 1) * filter[3];
        acc3 = get(idx + 1, idy - 1) * filter[7];

        acc1 += get(idx - 1, idy) * filter[1];
        acc2 += get(idx, idy) * filter[4];
        acc3 += get(idx + 1, idy) * filter[8];

        acc1 += get(idx - 1, idy + 1) * filter[2];
        acc2 += get(idx, idy + 1) * filter[5];
        acc3 += get(idx + 1, idy + 1) * filter[9];

        set(idx, idy, acc1);
    } else if (idx < width && idy < height) {
        if (idy > 0) {
            if (idx > 0)
                acc1 += get(idx - 1, idy - 1) * filter[0];
            acc1 += get(idx, idy - 1) * filter[3];
            if (idx < width)
                acc1 += get(idx + 1, idy - 1) * filter[7];
        }
        if (idx > 0)
            acc1 += get(idx - 1, idy) * filter[1];
        acc1 += get(idx, idy) * filter[4];
        if (idx < width)
            acc1 += get(idx + 1, idy) * filter[8];

        if (idy < height) {
            if (idx > 0)
                acc1 += get(idx - 1, idy + 1) * filter[2];
            acc1 += get(idx, idy + 1) * filter[5];
            if (idx < width)
                acc1 += get(idx + 1, idy + 1) * filter[9];
        }
        set(idx, idy, acc1);
    }
}

double conv3(float** data, unsigned long width, unsigned long height, const float* filter) {

    data_t *inData = *data;

    //defs
    float * cudaIn, *cudaOut, *cudaFilter;


    //Init cuda
    hipMalloc(&cudaIn, 0);

    tic();
    mlockall(0);
    hipMalloc(&cudaFilter, sizeof (float)*9);
    hipMalloc(&cudaIn, sizeof (float)*width * height);
    hipMalloc(&cudaOut, sizeof (float)*width * height);

    //Calcualte block
    dim3 block;
    block.x = blockx;
    block.y = blocky;
    block.z = 1;
    dim3 blocks;
    blocks.x = width / block.x + (int) ((width % block.x) > 0);
    blocks.y = height / block.y + (int) ((height % block.y) > 0);
    blocks.z = 1;
    //do
    hipMemcpy(cudaFilter, filter, sizeof (float)*9, hipMemcpyHostToDevice);
    hipMemcpy(cudaIn, inData, sizeof (float)*width*height, hipMemcpyHostToDevice);
    kernel_conv3 << <blocks, block>>>(cudaIn, cudaOut, width, height, cudaFilter);
    hipMemcpy(inData, cudaOut, sizeof (float)*width*height, hipMemcpyDeviceToHost);
    hipFree(cudaIn);
    hipFree(cudaOut);
    hipFree(cudaFilter);
    double time = toc();

    hipError_t hipError_t = hipGetLastError();
    printf("CudaError:\n%s\n", hipGetErrorString(hipError_t));

    return time;
}

__global__ void kernel_conv5(float * inData, float *outData, unsigned long width, unsigned long height, const float * filter) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;

    float acc1,acc2,acc3;
    acc1 = acc2 =acc3 = 0;

    if (idx < width - 2 && idy < height - 2 && idx > 1 && idy > 1) {
        for (int sx = -2; sx <= 2; sx++) {
            acc1 += get(idx + sx, idy -2) * filter[-2 * 5 + sx];
            acc2 += get(idx + sx, idy -1) * filter[-1 * 5 + sx];
            acc3 += get(idx + sx, idy + 0) * filter[0 * 5 + sx];
            acc1 += get(idx + sx, idy + 1) * filter[1 * 5 + sx];
            acc2 += get(idx + sx, idy + 2) * filter[2 * 5 + sx];
        }
        set(idx, idy, acc1 + acc2 + acc3);
    } else if (idx < width && idy < height) {
        for (int sy = -2; sy <= 2; sy++)
            for (int sx = -2; sx <= 2; sx++) {
                if (idx + sx > 0 && idx + sx < width && idx + sy > 0 && idx + sy < height)
                    acc1 += get(idx + sx, idy + sy) * filter[sy * 5 + sx];
            }
        set(idx, idy, acc1);
    }
}

double conv5(float** data, unsigned long width, unsigned long height, const float* filter) {

    data_t *inData = *data;

    //defs
    float * cudaIn, *cudaOut, *cudaFilter;


    //Init cuda
    hipMalloc(&cudaIn, 0);

    tic();
    mlockall(0);
    hipMalloc(&cudaFilter, sizeof (float)*25);
    hipMalloc(&cudaIn, sizeof (float)*width * height);
    hipMalloc(&cudaOut, sizeof (float)*width * height);

    //Calcualte block
    dim3 block;
    block.x = blockx;
    block.y = blocky;
    block.z = 1;
    dim3 blocks;
    blocks.x = width / block.x + (int) ((width % block.x) > 0);
    blocks.y = height / block.y + (int) ((height % block.y) > 0);
    block.z = 1;
    //do
    hipMemcpy(cudaFilter, filter, sizeof (float)*25, hipMemcpyHostToDevice);
    hipMemcpy(cudaIn, inData, sizeof (float)*width*height, hipMemcpyHostToDevice);
    kernel_conv5 << <blocks, block>>>(cudaIn, cudaOut, width, height, cudaFilter);
    hipMemcpy(inData, cudaOut, sizeof (float)*width*height, hipMemcpyDeviceToHost);
    hipFree(cudaIn);
    hipFree(cudaOut);
    hipFree(cudaFilter);
    double time = toc();

    hipError_t hipError_t = hipGetLastError();
    printf("CudaError:\n%s\n", hipGetErrorString(hipError_t));

    return time;
}