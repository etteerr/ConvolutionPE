#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <sys/mman.h>

#include "conv.h"
#include "tictoc.h"
#include "iaca.h"


#define get(X,Y) inData[(Y*width)+X]
#define set(X,Y,V) outData[(Y*width)+X] = V

__global__ void kernel_conv3(float * inData, float *outData, unsigned long width, unsigned long height, const float * filter) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x; 
    unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    float acc1;
    acc1=0;
    
    if (idx < width && idy < height) {
        //Valid target
        if (idy > 0) {
            if (idx > 0) 
                acc1 += get(idx-1, idy-1) * filter[0];
            acc1 +=     get(idx,idy-1) * filter[3];
            if (idx < width) 
                acc1 += get(idx+1, idy-1) * filter[7];
        }
        if (idx > 0) 
            acc1 += get(idx-1, idy) * filter[1];
        acc1 +=     get(idx,idy) * filter[4];
        if (idx < width) 
            acc1 += get(idx+1, idy) * filter[8];
        
        if (idy < height) {
            if (idx > 0) 
                acc1 += get(idx-1, idy+1) * filter[2];
            acc1 +=     get(idx,idy+1) * filter[5];
            if (idx < width) 
                acc1 += get(idx+1, idy+1) * filter[9];
        }
        set(idx,idy,acc1);
    }
}

double conv3(float** data, unsigned long width, unsigned long height, const float* filter) {

    data_t *inData = *data;
    
    //defs
    float * cudaIn, *cudaOut, *cudaFilter;
    
    
    //Init cuda
    hipMalloc(&cudaIn, 0);
    
    tic();
    mlockall(0);
    hipMalloc(&cudaFilter, sizeof(float)*9);
    hipMalloc(&cudaIn, sizeof(float)*width*height);
    hipMalloc(&cudaOut, sizeof(float)*width*height);
    
    //Calcualte block
    dim3 block;
    block.x = 8;
    block.y = 8;
    block.z = 0;
    dim3 blocks;
    blocks.x = width/8 + (int)((width%8)>0);
    blocks.y = height/8 + (int)((height%8)>0);
    blocks.z = 0;
    
    //do
    hipMemcpy(cudaFilter, filter, sizeof(float)*9,hipMemcpyHostToDevice);
    hipMemcpy(cudaIn, inData, sizeof(float)*width*height, hipMemcpyHostToDevice);
    kernel_conv3<<<block, blocks>>>(cudaIn, cudaOut, width, height, cudaFilter);
    hipMemcpy(inData, cudaOut, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipFree(cudaIn);
    hipFree(cudaOut);
    hipFree(cudaFilter);
    double time = toc();

    return time;
}

__global__ void kernel_conv5(float * inData, float *outData, unsigned long width, unsigned long height, const float * filter) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x; 
    unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    float acc1;
    acc1=0;
    
    if (idx < width && idy < height) {
        for(int sy=-2; sy <= 2; sy++)
            for(int sx=-2; sx <= 2; sx++) {
                if (idx+sx > 0 && idx+sx < width && idx+sy>0 && idx+sy<height) 
                    acc1 += get(idx+sx, idx+sy) * filter[sy*5+sx];
            }
        set(idx,idy,acc1);
    }
}

double conv5(float** data, unsigned long width, unsigned long height, const float* filter) {

    data_t *inData = *data;
    
    //defs
    float * cudaIn, *cudaOut, *cudaFilter;
    
    
    //Init cuda
    hipMalloc(&cudaIn, 0);
    
    tic();
    mlockall(0);
    hipMalloc(&cudaFilter, sizeof(float)*25);
    hipMalloc(&cudaIn, sizeof(float)*width*height);
    hipMalloc(&cudaOut, sizeof(float)*width*height);
    
    //Calcualte block
    dim3 block;
    block.x = 8;
    block.y = 8;
    dim3 blocks;
    blocks.x = width/8 + (width%8)>0;
    blocks.y = height/8 + (height%8)>0;
    
    //do
    hipMemcpy(cudaFilter, filter, sizeof(float)*25,hipMemcpyHostToDevice);
    hipMemcpy(cudaIn, inData, sizeof(float)*width*height, hipMemcpyHostToDevice);
    kernel_conv5<<<block, blocks>>>(cudaIn, cudaOut, width, height, cudaFilter);
    hipMemcpy(inData, cudaOut, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipFree(cudaIn);
    hipFree(cudaOut);
    hipFree(cudaFilter);
    double time = toc();

    return time;
}